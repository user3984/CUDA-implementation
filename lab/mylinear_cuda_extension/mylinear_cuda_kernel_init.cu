#include "hip/hip_runtime.h"
// MIT License

// Copyright (c) Microsoft Corporation.

// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:

// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.

// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE

#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

template <typename scalar_t>
__global__ void matmul_kernel(
    const scalar_t* A,
    const scalar_t* B,
    scalar_t* C,
    const int M, 
    const int K, 
    const int N,
    const bool trans_A = false,
    const bool trans_B = false) 
{
    const int row = blockIdx.x * blockDim.x + threadIdx.x;
    const int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < M && col < N)
    {
        scalar_t sum = 0.0;
        for (int k = 0; k < K; k++)
        {
            const int i = trans_A ? (k * M + row) : (row * K + k);
            const int j = trans_B ? (col * K + k) : (k * N + col);
            sum += A[i] * B[j];
        }

        C[row * N + col]  = sum;
    }
}

std::vector<torch::Tensor> mylinear_cuda_forward(
    torch::Tensor input,
    torch::Tensor weights)
{
    const int M = input.size(0);
    const int K = input.size(1);
    const int N = weights.size(0);

    auto output = torch::zeros({M, N}, torch::TensorOptions().device(torch::kCUDA));

    const dim3 block(32, 32);
    const dim3 grid((M - 1) / 32 + 1, (N - 1) / 32 + 1);

    AT_DISPATCH_FLOATING_TYPES(input.type(), "mylinear_cuda_forward", ([&] {
        matmul_kernel<scalar_t><<<grid, block>>>(
            input.data<scalar_t>(),
            weights.data<scalar_t>(),
            output.data<scalar_t>(),
            M,
            K,
            N,
            false,
            true);
        }));
    
    return {output};
}

std::vector<torch::Tensor> mylinear_cuda_backward(
    torch::Tensor grad_output,
    torch::Tensor input,
    torch::Tensor weights)
{
    const int M = grad_output.size(0);
    const int N = grad_output.size(1);
    const int K = weights.size(1);

    auto grad_input = torch::zeros({M, K}, torch::TensorOptions().device(torch::kCUDA));
    auto grad_weights = torch::zeros({N, K}, torch::TensorOptions().device(torch::kCUDA));

    const dim3 block(32, 32);
    const dim3 grid1((M - 1) / 32 + 1, (K - 1) / 32 + 1);
    const dim3 grid2((N - 1) / 32 + 1, (K - 1) / 32 + 1);


    AT_DISPATCH_FLOATING_TYPES(input.type(), "mylinear_cuda_backward_input", ([&] {
        matmul_kernel<scalar_t><<<grid1, block>>>(
            grad_output.data<scalar_t>(),
            weights.data<scalar_t>(),
            grad_input.data<scalar_t>(),
            M,
            N,
            K,
            false,
            false);
        }));

    AT_DISPATCH_FLOATING_TYPES(input.type(), "mylinear_cuda_backward_input", ([&] {
        matmul_kernel<scalar_t><<<grid2, block>>>(
            grad_output.data<scalar_t>(),
            input.data<scalar_t>(),
            grad_weights.data<scalar_t>(),
            N,
            M,
            K,
            true,
            false);
        }));
    
    return {grad_input, grad_weights};
}